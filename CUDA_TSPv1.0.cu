//By: Goodwyn, Zach & Mueller, Jerome
//This program uses a CUDA-enabled GPU to brute-force solve the travelling salesman problem


#include <hip/hip_runtime.h>
#include<stdio.h>
#include"math.h"
#include<stdlib.h>
#include<time.h>
#include<sys/time.h> // Used for timing this.
#include<unistd.h> // Used for timing this.
#include<algorithm>//needed for next_permutation
#include<climits>//needed for MAX_INT
#include<assert.h>
#include<time.h>
#include<sys/time.h> // Used for timing this.
#include<unistd.h> // Used for timing this.

#define MAX_PATH 1000
//#define NUM_CORES 96

__host__ __device__ void convertFact(unsigned long long, int*);
__host__ __device__ void orderPermutation(int*, unsigned long long, int);
unsigned long long factorial(unsigned long long);
int* generateArray(int num_Cities);
int charToInt(char* temp_Input);
void checkCuda(hipError_t problem, int id);
__global__ void permute(int* device_Matrix, short* device_Cities, long long* device_Perm_Start_Indecies, int* device_Least_Cost_Array,long long* device_Least_Path_Array, int num_Cities, int threads_Per_Block);

//returns the factorial for the given number
//x: the index for which factorial number you want.
//returns: x!
unsigned long long factorial(unsigned long long x)
{
	if(x == 1)
		return 1;
	else
		return x * factorial(x-1);
}

//converts a number into base factoriaint** adjacency_Matrix = new int*[num_Cities];
//num: the number to convert to base factorial
//digits: a storage array to store the digits of the base factorial number
//numbers are stored in reverse order (so the 2nd digit is in position 1, the third in 2, etc.
//digits[0] will contain the length of the number in digits, since the first number of a base factorial number is always 0
__host__ __device__ void convertFact(unsigned long long num, int* digits)
{
	int numDigits = 1;//there is always a spare 0 in these numbers
	while(num > 0)
	{
		digits[numDigits] = num % (numDigits + 1);
		numDigits++;
		num /= numDigits;
	}
	digits[0] = numDigits;//because the first digit is always zero, we will 	store the length of the array in the 0th slot
}

//returns the array transposed to the nth permutation after the given ordering
//toOrder: the set for which you would like to define the nth permutation
//m: the numbered permutation to be stored in toOrder
//size: the size of the array toOrder
//warning: gives unpredictable behavior if n is > the number of unique permutations for toOrder
__host__ __device__ void orderPermutation(short* toOrder, unsigned long long n, int size)
{
	int swaps[100];
	convertFact(n, swaps);
	int location = size - swaps[0];//accounts for leading zeros
	int loc = swaps[0] - 1;//used to iterate through the digits of the 					factoradic number
	while(loc > 0)
	{
		int temp = toOrder[location + swaps[loc]];
		for(int x = location+swaps[loc]; x > location; x--)		
		{
			toOrder[x] = toOrder[x - 1];
		}
		toOrder[location] = temp;
		location++;
		loc--;
	}

}

//returns the array transposed to the nth permutation after the given ordering
//toOrder: the set for which you would like to define the nth permutation
//m: the numbered permutation to be stored in toOrder
//size: the size of the array toOrder
//warning: gives unpredictable behavior if n is > the number of unique permutations for toOrder
__host__ __device__ void orderPermutation(int* toOrder, unsigned long long n, int size)
{
	int swaps[100];
	convertFact(n, swaps);
	int location = size - swaps[0];//accounts for leading zeros
	int loc = swaps[0] - 1;//used to iterate through the digits of the 					factoradic number
	while(loc > 0)
	{
		int temp = toOrder[location + swaps[loc]];
		for(int x = location+swaps[loc]; x > location; x--)		
		{
			toOrder[x] = toOrder[x - 1];
		}
		toOrder[location] = temp;
		location++;
		loc--;
	}

}

//returns a pointer to a "two" dimensional randomly generated symmetrical adjacency matrix
//num_Cities: used to decide how much memory should be allocated
int* generateArray(int num_Cities)
{
	int* temp_Matrix = (int*)malloc(num_Cities*num_Cities*sizeof(int*));

	/*for(int index = 0; index < num_Cities; index++)
	{
		temp_Matrix[index] = (int*)malloc(num_Cities*sizeof(int));
	}*/
	for(int outer = 0; outer < num_Cities; outer++)
	{

		for(int inner = 0; inner <= outer; inner++)
		{

			temp_Matrix[outer * num_Cities + inner] = 1 + rand() % MAX_PATH;

			temp_Matrix[inner * num_Cities + outer] = temp_Matrix[outer * num_Cities + inner];
		}

	}

	return temp_Matrix;
	

}

//Converts a character string to an integer
//temp_Input: A cstring (char*) containing a number to translate
//Gives unpredictable, but not undefined behavior if temp_Input contains non-numeric characters,
//or if temp_Input is too large to be held in an int. Does not support negatives or decimals
int charToInt(char* temp_Input)
{
	int num_Digit = 0;
	int number = 0;

	while(temp_Input[num_Digit] != '\0')//loops until the end of the string
	{
		number = number * 10;//accounts for the place-value of the digits
		if(temp_Input[num_Digit] != '0')
		{
			//in character sets, the character one minus '1' yields zero, '2' minus '1' yields 1, etc.
			number += temp_Input[num_Digit] + 1 - '1';
		}
		num_Digit++;
	}
	return number;
		
}

//shorthand to check if a cuda error occured
//problem: cudaError_t storing the result of a cudaMalloc
//id: an integer id that gets print out with the message, default = -1
void checkCuda(hipError_t problem, int id = -1)
{
	if(problem != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(problem));
		printf("Terminating process with id = %d\n", id);
		abort();
	}
}

//Kernel Function!
//Each thread calculates its starting and ending index, then calculates the total costs of the paths assigned to it, then stores its best result in the device_Least_Cost_Array, and the index of that path in the device_Least_Path_Array
//ALL ARGUMENTS POINT TO THE GPU'S COPY OF THE GIVEN DATA STRUCTURE
//device_Matrix: The adjacency matrix representing the different costs of getting from one city to another
//device_Cities: The array containing EVERY thread's representation of the cities (i.e. each thread gets an array of num_Cities elements containing numbers 1 - [num_Cities-1]) See comments in method body for further comments
//device_Perm_Start_Indecies: The starting locations for a given Block of threads. It is up to the individual thread to calculate which paths to run based on this array and its thread ID
//device_Least_Cost_Array: When a thread has finished its permutations, it writes the cost of the shortest path it found into a unique slot in this array
//device_Least_Path_Array: The corresponding permutation number for the shortest calculated path. Used to retrieve the city-ordering for the best solution
//num_Cities: The number of cities in this instance of the travelling salesman problem
//threads_Per_Block: The number of threads in a given block. 
__global__ void permute(int* device_Matrix, short* device_Cities, long long* device_Perm_Start_Indecies, int* device_Least_Cost_Array,long long* device_Least_Path_Array, int num_Cities, int threads_Per_Block)
{
	int id = blockIdx.x * threads_Per_Block + threadIdx.x;//this id is unique for each thread
	//each thread works with a subset of the device_Cities array, this next statement procures the starting address of this thread's subset
	short* cities = device_Cities + (id * num_Cities);
	
	long long index;//which path permutation the thread is calculating
	int tot_Sum;//the running cost of the permutation this thread is calculating
	long long least_Path = 0;//the permutation id# of the shortest path this thread has found
	int least_Cost = 999999;//the least cost solution this thread has so far found
	float sectionRatio = (float)1.0/threads_Per_Block;//calculates what portion of the thread's block's workload needs to be completed by this thread
	long long block_Perms = device_Perm_Start_Indecies[blockIdx.x + 1] - device_Perm_Start_Indecies[blockIdx.x];//the total number permutations in this thread's block's workload
	long long start_Perm = (sectionRatio * threadIdx.x) * block_Perms;//An offset denoting which path permutation number this thread should start to calculate
	long long end_Perm = (sectionRatio * (threadIdx.x + 1)) * block_Perms;//An offset denoting the one permutation beyond what this thread should calculate

	orderPermutation(cities, device_Perm_Start_Indecies[blockIdx.x] + start_Perm, num_Cities);//initializes this thread's cities array to the appropriate ordering	

	//loops through all the permutations assigned to this thread
	for(index = device_Perm_Start_Indecies[blockIdx.x] + start_Perm ; index < device_Perm_Start_Indecies[blockIdx.x] + end_Perm ; index++)
	{
		tot_Sum = 0;
		int inner;
		for(inner = 0; inner < num_Cities; inner++)//for each city, looks up the distance to the next city and adds it to a running sum
		{
			tot_Sum += device_Matrix[cities[inner] * num_Cities + cities[(inner + 1) % num_Cities]];	
		}

		if(tot_Sum < least_Cost)//updates if the soplution is the best so far
		{
			least_Cost = tot_Sum;
			least_Path = index;
		}
		for(inner = 0; inner < num_Cities; inner++)//resets the cities array for use in orderPermutation
		{
			cities[inner] = inner;
		}
		orderPermutation(cities, index + 1, num_Cities);//sets the cities array to the next permutation
	}
	//writes this thread's best solutions to the two arrays for transfer back to the host
	device_Least_Cost_Array[id] = least_Cost;
	device_Least_Path_Array[id] = least_Path;	

	
}



int main(int argc, char* argv[])
{
	//initialize timer
	struct timeval startTime, endTime;
	gettimeofday(&startTime, NULL);

	//variables corresponding to the arguments
	unsigned int seeder;
	int num_Threads;//(threads per block)
	int num_Blocks;
	int num_Cities;
	int total_Threads;


	if(argc != 5)//if an improper number of parameters were passed
	{
		printf("Error: improper number of commands");
		printf("arguments: #cities (seed) (requestedCores)");
		fflush(stdout);
	}

	num_Cities = charToInt(argv[1]);

	seeder = charToInt(argv[2]);
	srand(seeder);

	num_Blocks = charToInt(argv[3]);
	
	num_Threads = charToInt(argv[4]);

	total_Threads = num_Blocks * num_Threads;
	
	//calculates the starting index for each block
	double section_Ratio = double(1)/num_Blocks;

	long long total_Perms = factorial(num_Cities);

	long long* perm_Start_Indecies = (long long*)malloc((num_Blocks + 1) * sizeof(long long));

	int index;
	for(index = 0; index < num_Blocks + 1; index++)
	{	
		perm_Start_Indecies[index] = total_Perms * (section_Ratio * index);
	//	printf("%d index %lld\n", index, perm_Start_Indecies[index]);
	//	fflush(stdout);
	}

	//Following section allocates memory on the host and on the device, and transfers the adjacency matrix the cities array, and the starting index array to the device
	hipError_t problemo;

	long long* device_Perm_Start_Indecies;
	problemo = hipMalloc((void**)&device_Perm_Start_Indecies, sizeof(long long) * (num_Blocks + 1));
	checkCuda(problemo,1);
	problemo = hipMemcpy(device_Perm_Start_Indecies, perm_Start_Indecies, (sizeof(long long) * (num_Blocks + 1)), hipMemcpyHostToDevice);
	checkCuda(problemo,2);

	int* adjacency_Matrix = generateArray(num_Cities);
	
/*	int foo = 0;
	for(foo; foo < (num_Cities * num_Cities); foo++)
	{
		printf("%d\t",adjacency_Matrix[foo]);
		if((foo + 1) % num_Cities == 0)
		{
			printf("\n");
		}
	}
*/	
	int* device_Matrix;
	problemo = hipMalloc((void**)&device_Matrix, num_Cities*num_Cities*sizeof(int));
	checkCuda(problemo,3);
	problemo = hipMemcpy(device_Matrix, adjacency_Matrix,num_Cities*num_Cities*sizeof(int),hipMemcpyHostToDevice);
	checkCuda(problemo,4);

	int* device_Least_Cost_Array;
	problemo = hipMalloc((void**)&device_Least_Cost_Array, total_Threads * sizeof(int));
	checkCuda(problemo,5);

	long long* device_Least_Path_Array;
	problemo = hipMalloc((void**)&device_Least_Path_Array, total_Threads * sizeof(long long));
	checkCuda(problemo,6);

	short* cities = (short*)malloc(num_Cities * total_Threads * sizeof(short));

	for(index = 0; index < total_Threads; index++)//initializes the cities array with the appropriate values
	{
		int inner = 0;
		for(inner = 0; inner < num_Cities; inner++)
		{	
			cities[index * num_Cities + inner] = inner;
		}
	}

	short* device_Cities;
	problemo = hipMalloc((void**)&device_Cities, num_Cities * total_Threads * sizeof(short));
	checkCuda(problemo,7);

	problemo = hipMemcpy(device_Cities, cities, num_Cities * total_Threads * sizeof(short), hipMemcpyHostToDevice);
	checkCuda(problemo,8);

	int* least_Cost_Array = (int*)malloc(total_Threads * sizeof(int));
	
	long long* least_Path_Array = (long long*)malloc(total_Threads * sizeof(long long));

	for(index = 0; index < total_Threads; index++)
	{
		least_Cost_Array[index] = 2;
		least_Path_Array[index] = 2;
	}

	//printf("fertig!1\n");
	//fflush(stdout);

	
	//kernel call//////////////////////////////////////////////////////////////////////////////////

	permute<<<num_Blocks,num_Threads>>>(device_Matrix, device_Cities, device_Perm_Start_Indecies, device_Least_Cost_Array, device_Least_Path_Array, num_Cities, num_Threads);
	checkCuda(hipGetLastError(),13);


	//printf("fertig!2\n");
	//fflush(stdout);
	
	//retrieves the arrays storing the best results from each threas
	problemo = hipMemcpy(least_Cost_Array, device_Least_Cost_Array,total_Threads * sizeof(int), hipMemcpyDeviceToHost);
	checkCuda(problemo,9);
	problemo = hipMemcpy(least_Path_Array, device_Least_Path_Array,total_Threads * sizeof(long long), hipMemcpyDeviceToHost);
	checkCuda(problemo,10);	

	//initializes an int[] to store the cities of the best path
	int* true_Cities = (int*)malloc(num_Cities*sizeof(int));
	index = 0;
	for(index = 0; index < num_Cities; index++)
	{
		true_Cities[index] = index;
	}	

	int block_Index;
	int temp_Best = 99999999;
	int best_Index = 0;
	
	//calculates the best path of those returned by the GPU
	for(block_Index = 0; block_Index < total_Threads; block_Index++)
	{
		//printf("%d << leastCost element %d\n",least_Cost_Array[block_Index], block_Index);
		if(least_Cost_Array[block_Index] < temp_Best)
		{
			best_Index = block_Index;
			temp_Best = least_Cost_Array[block_Index];
		}
	}

	//displays the results
	printf("%d  << best! from thread %d\n",temp_Best, best_Index);
	orderPermutation(true_Cities, least_Path_Array[best_Index], num_Cities);
	index = 0;
	for(index = 0; index < num_Cities; index++)
	{
		printf("%d\t", true_Cities[index]);
	}

	printf("\nFinished!\n");
	//system("PAUSE");
	// Timing code adapted from: http://stackoverflow.com/questions/588307/c-obtaining-milliseconds-time-on-linux-clock-doesnt-seem-to-work-properl
	gettimeofday(&endTime, NULL);
	long timeDelta, startSeconds, startUSeconds, stopSeconds, stopUSeconds, startTotal, stopTotal;
	startSeconds = startTime.tv_sec;
	stopSeconds = endTime.tv_sec;
	startUSeconds = startTime.tv_usec;
	stopUSeconds = endTime.tv_usec;
	startTotal = (startSeconds * 1000) + (startUSeconds / 1000);
	stopTotal = (stopSeconds * 1000) + (stopUSeconds / 1000);


	timeDelta = stopTotal - startTotal;
	printf("Time: %d milliseconds\n",timeDelta);

}















