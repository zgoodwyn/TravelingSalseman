#include<stdlib.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
//tehnically unnecessary

//indexes "threadIdx.x" elements into the array, adds threadId.x to it (effectively doubling it), then adding 1
__global__ void multiGo(float* arr)
{
	arr[threadIdx.x] += threadIdx.x + 1;
}

int main()
{
	int N = 5;
	size_t size = N * sizeof(float);//the size in bytes of all data transfers, used in cuda-specific calls and mallocs

	float* h_A = (float*)malloc(size);
	float* h_B = (float*)malloc(size);

	int index = 0;

	//fills h_A with the numbers 0-(n-1)
	for(index = 0; index < N; index = index + 1)
	{
		h_A[index] = index;
		printf("%f\n",h_A[index]);
	}

	float* d_A;
	hipError_t problemo;//used for error checking, optional (cudamalloc returns a cudaError_t)
	problemo = hipMalloc((void**)&d_A, size);
	if(problemo != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(problemo));
	}
	//copies the contents of h_A (on the host) into d_A (on the device/GPU)
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	int id = 0;
	//initializes elements of h_B to -1
	for(id = 0; id < N; id = id + 1)
	{
		h_B[id] = -1;
		printf("%f\n",h_B[id]);
	}
	
	multiGo<<<1,5>>>(d_A);//Cuda function call
	
	//copies the cuda memory back into the h_B on the device
	hipMemcpy(h_B, d_A, size, hipMemcpyDeviceToHost);

	//prints out the elements of B
	int index_Two = 0;
	for(index_Two = 0; index_Two < N; index_Two = index_Two + 1)
	{
		printf("%f\n", h_B[index_Two]);
	}

	free(h_A);
	hipFree(d_A);

}
