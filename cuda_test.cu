#include<stdlib.h>
#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void multiGo(float* arr)
{
	arr[threadIdx.x] += threadIdx.x + 1;
}

int main()
{
	int N = 5;
	size_t size = N * sizeof(float);

	float* h_A = (float*)malloc(size);
	float* h_B = (float*)malloc(size);

	int index = 0;

	for(index = 0; index < N; index = index + 1)
	{
		h_A[index] = index;
		printf("%f\n",h_A[index]);
	}

	float* d_A;
	hipError_t problemo;
	printf("here\n");
	fflush(stdout);
	problemo = hipMalloc((void**)&d_A, size);
	if(problemo != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(problemo));
	}

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	int id = 0;
	for(id = 0; id < N; id = id + 1)
	{
		h_B[id] = -1;
		printf("%f\n",h_B[id]);
	}
	
	multiGo<<<1,5>>>(d_A);
	
	hipMemcpy(h_B, d_A, size, hipMemcpyDeviceToHost);

	int index_Two = 0;
	for(index_Two = 0; index_Two < N; index_Two = index_Two + 1)
	{
		printf("%f\n", h_B[index_Two]);
	}

	free(h_A);
	hipFree(d_A);

}
